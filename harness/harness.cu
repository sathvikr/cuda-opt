#include <iostream>
#include <hip/hip_runtime.h>

// Kernel function declaration
__global__ void matmulKernel(float* A, float* B, float* C, int N);

int main() {
    int N = 1024; // Example size, can be adjusted
    size_t size = N * N * sizeof(float);

    // Allocate host memory
    float* h_A = (float*)malloc(size);
    float* h_B = (float*)malloc(size);
    float* h_C = (float*)malloc(size);

    // Initialize host matrices
    for (int i = 0; i < N * N; ++i) {
        h_A[i] = static_cast<float>(rand()) / RAND_MAX;
        h_B[i] = static_cast<float>(rand()) / RAND_MAX;
    }

    // Allocate device memory
    float* d_A;
    float* d_B;
    float* d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    // Copy data from host to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Define block and grid dimensions
    dim3 blockDim(16, 16);
    dim3 gridDim((N + blockDim.x - 1) / blockDim.x, 
                 (N + blockDim.y - 1) / blockDim.y);

    // Launch the kernel
    matmulKernel<<<gridDim, blockDim>>>(d_A, d_B, d_C, N);

    // Copy result from device to host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Verify the result (optional)
    // for (int i = 0; i < N * N; ++i) {
    //     std::cout << h_C[i] << " ";
    //     if ((i + 1) % N == 0) std::cout << std::endl;
    // }

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}
